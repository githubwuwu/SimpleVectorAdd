#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <string>
#include <vector>
#include <map>
#include <iostream>
#include <sstream>
#include "hip/device_functions.h"
#include <hip/hip_runtime_api.h>
#include <fstream>
using std::vector;
using std::string;
using std::map;

#define THREAD_NUM 512
#define BLOCK_NUM 1024
const float pi = 3.1415926;

__constant__ static int dd_angle[30];
__constant__ static float dd_start[30];
__constant__ static float d_s_angle_1[30];
__constant__ static float d_s_angle_2[30];


struct InitData   //��������������
{
	InitData(int c, int fs, int f0, float width, float kerf, int N_elements, int length, float image_length, vector<int> angle) :
		speed(c), sample_frequency(fs), central_frequency(f0), width(width), kerf(kerf), N_elements(N_elements), data_length(length), image_length(image_length), angle(angle)
	{
		pitch = width + kerf;
		array_length = pitch*(N_elements - 1) + width;
		d_x = array_length / N_elements;
		d_z = double(1) / fs;
	}

	void push_tstart(float tstart, int i)
	{
		tstatrt[i] = tstart;
	}

	//ԭʼ����
	int speed;
	float sample_frequency;
	int central_frequency;
	float width;
	float kerf;
	int N_elements;
	int data_length;
	float pitch;
	float array_length;
	float d_x;
	float image_length;
	double d_z;
	vector<int> angle;
	map<int, float> tstatrt;
};

// analyse��readData�Ƕ�ȡdat�ļ��ĺ���
void analyse(float* in, const char* buf)
{
	string contents = buf;
	string::size_type pos1 = 0;
	int n = 0;
	int i = 0;
	while ((pos1 = contents.find_first_of("+-.0123456789e", pos1)) != string::npos)
	{
		auto pos2 = contents.find_first_not_of("+-.0123456789e", pos1);
		n = pos2 - pos1;
		float d = stod(contents.substr(pos1, n));
		in[i++] = d;
		pos1 += n;
	}
}

float* readData(string path, InitData &init)
{
	int one_frame_length = init.N_elements*init.data_length;
	int all_data_length = (init.angle.size())*one_frame_length;
	float *all_rf = new float[all_data_length];
	float *t_start = new float[init.angle.size()];

	const int MAXS = one_frame_length * 20;//�����ַ�����
	char *buf = new char[MAXS];
	char *t_buf = new char[20];
	int kk = 0;

	for (auto ii : init.angle)
	{
		std::cout << "���ڶ�ȡ��" << ii << "֡����" << std::endl;

		std::stringstream pathname;
		pathname << ii;
		string file_path_now = path + "data_" + pathname.str() + ".dat";
		std::ifstream ifs(file_path_now, std::ios::binary);
		if (ifs)
		{
			float *data = all_rf + one_frame_length*kk;
			ifs.read((char*)data, one_frame_length*sizeof(data));

		}

		string t_path = path + "tstart_" + pathname.str() + ".txt";
		const char* t_file_path = t_path.c_str();
		FILE* t_fp = fopen(t_file_path, "rb");

		if (t_fp)
		{
			int len = fread(t_buf, 1, 20, t_fp);
			t_buf[len] = '\0';
			analyse(t_start + kk, t_buf);//��������
			init.push_tstart(t_start[kk], ii);
		}
		kk++;
	}
	delete buf;
	return all_rf;

}

//��һ�����м���ĺ˺���
__global__ void cuda_compoundData(float* out, float* in, int new_length, int length, int N_elements, float pitch, int angle_n,
	int fs, int c, double d_z)
{
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int row = blockIdx.y*blockDim.y + threadIdx.y;

	int angle_index = row / N_elements;
	float t1 = dd_start[angle_index];
	float temp1 = d_s_angle_1[angle_index];
	float temp2 = d_s_angle_2[angle_index];
	int real_row = row - angle_index*N_elements;
	float i_real_own = (dd_angle[angle_index] > 0) ? real_row*pitch : (N_elements - real_row - 1)*pitch;

	float j_real = d_z*(col + 1) *c / 2;

	//��������ѭ������ļ�����
	float j_real_2 = j_real*j_real;
	int oneFrameLength = N_elements*new_length;
	float j_temp1 = j_real*temp1;
	float i_temp2 = i_real_own*temp2;
	for (int row_i = 0; row_i != N_elements; ++row_i)
	{
		float i_real = (real_row - row_i)*pitch;
		int jj = ((j_temp1 + i_temp2 + (sqrtf(j_real_2 + i_real *i_real))) / c - t1)*fs - 0.5f;//ȷ��������������
		if ((jj >= 0) && (jj < new_length))
		{
			out[row*new_length + col] += in[angle_index*oneFrameLength + row_i*new_length + jj];
		}
	}
}

//�ڶ������м���ĺ˺���
__global__ void cuda_AddData(float* out, float* in, int length, int N_elements, int angle_n)
{
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int row = blockIdx.y*blockDim.y + threadIdx.y;

	for (int i = 0; i != angle_n; ++i)
	{
		out[row*length + col] += in[i*N_elements*length + row*length + col];
	}
}

//���ú˺�������Ҫ���㺯��
void compoundData(float* data, InitData &init)
{
	//�������ļ���
	int one_frame_length = init.N_elements*init.data_length;
	int all_data_length = (init.angle.size())*one_frame_length;

	int new_length = (init.data_length / 32 + 1) * 32;//�߳����ı���
													  //int new_length = 64;
	int new_one_frame_length = init.N_elements*new_length;
	int new_all_data_length = (init.angle.size())*new_one_frame_length;


	std::cout << "������ݳ���Ϊ" << new_length;

	//��ʱ����
	hipEvent_t startMemcpy; hipEvent_t stopMemcpy;
	hipEvent_t startKernel; hipEvent_t stopKernel;

	hipEventCreate(&startMemcpy);
	hipEventCreate(&stopMemcpy);
	hipEventCreate(&startKernel);
	hipEventCreate(&stopKernel);

	hipEventRecord(startMemcpy);  //����GPU�и��ƺͿ�����������ʱ��

	float *new_data = new float[new_all_data_length]();//������� �������豸�ڴ�Ŀ���
	for (int kk = 0; kk != init.angle.size(); ++kk)
	{
		for (int jj = 0; jj < init.data_length; jj++) {
			for (int ii = 0; ii < init.N_elements; ii++) {
				new_data[kk*new_one_frame_length + ii*new_length + jj] = data[kk*one_frame_length + ii*init.data_length + jj];
			}
		}
	}

	//����device�ڴ�
	float *d_new_data;
	hipMalloc(&d_new_data, new_all_data_length*sizeof(float));
	hipMemcpy(d_new_data, new_data, sizeof(float) * new_all_data_length, hipMemcpyHostToDevice);

	float *d_ans_data;                                    //�������� �豸�ڴ�Ŀ��ٺ͸�ֵ
	hipMalloc(&d_ans_data, new_one_frame_length*sizeof(float));
	float *ans_data = new float[new_one_frame_length]();


	const size_t smemSize = THREAD_NUM*sizeof(float);

	//�����ڴ� ��ŽǶȵ�����
	int *d_angle = new int[init.angle.size()];
	float *d_start = new float[init.angle.size()];
	float *s_angle_1 = new float[init.angle.size()];
	float *s_angle_2 = new float[init.angle.size()];
	for (int i = 0; i != init.angle.size(); ++i)
	{
		d_angle[i] = init.angle[i];
		d_start[i] = init.tstatrt[init.angle[i]];
		s_angle_1[i] = cos(float(d_angle[i])*pi / 180);
		s_angle_2[i] = sin(float(d_angle[i])*pi / 180);
	}

	//���ó����ڴ�
	hipMemcpyToSymbol(HIP_SYMBOL(dd_angle), d_angle, sizeof(int) * init.angle.size());
	hipMemcpyToSymbol(HIP_SYMBOL(dd_start), d_start, sizeof(float) * init.angle.size());
	hipMemcpyToSymbol(HIP_SYMBOL(d_s_angle_1), s_angle_1, sizeof(float) * init.angle.size());
	hipMemcpyToSymbol(HIP_SYMBOL(d_s_angle_2), s_angle_2, sizeof(float) * init.angle.size());

	checkCudaErrors(hipPeekAtLastError());
	checkCudaErrors(hipDeviceSynchronize());

	hipEventRecord(stopMemcpy);

	hipEventRecord(startKernel);//����˺�����ʱ

	float *d_temp_data;                                    //�������� �豸�ڴ�Ŀ��ٺ͸�ֵ
	hipMalloc(&d_temp_data, new_all_data_length*sizeof(float));
	float *temp_data = new float[new_all_data_length]();
	
	//���õ�һ���˺���
	dim3 dimBlock(8, 8, 1);
	dim3 dimGrid((new_length + dimBlock.x - 1) / dimBlock.x,
		(init.N_elements*init.angle.size() + dimBlock.y - 1) / dimBlock.y, 1);

	cuda_compoundData << <dimGrid, dimBlock >> >(d_temp_data, d_new_data, new_length, init.data_length, init.N_elements, init.pitch, init.angle.size(),
		init.sample_frequency, init.speed, init.d_z);
	checkCudaErrors(hipPeekAtLastError());
	checkCudaErrors(hipDeviceSynchronize());

	//���õڶ����˺���
	dim3 dimBlock2(8, 8, 1);
	dim3 dimGrid2((new_length + dimBlock.x - 1) / dimBlock.x,
		(init.N_elements + dimBlock.y - 1) / dimBlock.y, 1);

	cuda_AddData << <dimGrid2, dimBlock2 >> >(d_ans_data, d_temp_data, new_length, init.N_elements, init.angle.size());
	hipEventRecord(stopKernel);

	checkCudaErrors(hipPeekAtLastError());
	checkCudaErrors(hipDeviceSynchronize());

	//����device�����host
	hipMemcpy(temp_data, d_temp_data, sizeof(float) * new_all_data_length, hipMemcpyDeviceToHost);
	hipMemcpy(ans_data, d_ans_data, sizeof(float) * new_one_frame_length, hipMemcpyDeviceToHost);

	//����host��device�ڴ�
	hipFree(dd_angle);
	hipFree(dd_start);
	hipFree(d_s_angle_1);
	hipFree(d_s_angle_2);
	hipFree(d_ans_data);
	hipFree(d_new_data);

	delete d_angle;
	delete d_start;
	delete s_angle_1;
	delete s_angle_2;
	delete data;
	delete new_data;
	delete temp_data;
	delete ans_data;

	//������ʱ
	float memcpyTime = 0;
	hipEventElapsedTime(&memcpyTime, startMemcpy, stopMemcpy);
	float kernelTime = 0;
	hipEventElapsedTime(&kernelTime, startKernel, stopKernel);

	std::cout << "GPU��CPU�и���������ʱ" << memcpyTime << "ms" << std::endl;
	std::cout << "�˺���������ʱ����Ƕȸ��ϣ�" << kernelTime << "ms" << std::endl;

}

int main()
{
	string path = "..//data//";
	vector<int> angle = { -9,-7,-5,-3,-1, 0,1,3,5,7,9 };
	//vector<int> angle = { -7,-5,-3,-1, 0,1,3,5,7 };

	InitData init(1540, 50e6, 3.5e6, 0.2798e-3, 0.025e-3, 128, 6000, 0.11, angle);

	//std::cout << cos(1*pi/180);
	float* test = readData(path, init);

	compoundData(test, init);
	//std::cout << *test;
	std::cout << "���������";
}